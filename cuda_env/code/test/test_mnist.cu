#include <matrix.hpp>
#include <hiprand.h>
#include <hiprand/hiprand_kernel.h>
#include <math.h>
#include <layer.hpp>
#include <fstream>

#include <fstream>
#include <string>
#include <sstream>
#include <vector>

using namespace cuda_Matrix;

struct TwoLayerNetwork
{
    Affine Affine1;
    RelU   RelU1;
    Affine Affine2;

    SoftMaxWithLoss SoftMaxWithLoss1;
    bool onGpu;
    TwoLayerNetwork(int _input_size, int _hidden_size, int _batch_size, int _output_size, float weight_init_std=0.01, bool _onGpu = true):
    Affine1(_input_size, _hidden_size, _batch_size, weight_init_std, _onGpu),
    RelU1(_hidden_size, _batch_size, _onGpu),
    Affine2(_hidden_size, _output_size, _batch_size, weight_init_std, _onGpu),
    SoftMaxWithLoss1(_output_size, _batch_size, _onGpu)
    {

    }

    Matrix predict(const Matrix& data)
    {
        Matrix result = Affine2.forward(RelU1.forward(Affine1.forward(data)));
        return result;
    }

    Matrix loss(const Matrix& data, const Matrix& teacher)
    {
        Matrix test = SoftMaxWithLoss1.forward(predict(data), teacher);
        return SoftMaxWithLoss1.forward(predict(data), teacher);
    }

    void update_weight_bias(const Matrix& x, const Matrix& t, float learning_rate)
    {
        //forward
        loss(x, t);

        //backward
        Affine1.backward(RelU1.backward(Affine2.backward(SoftMaxWithLoss1.backward())));

        //update weight and bias
        Affine1.weight = Matrix::Add(Affine1.weight, Matrix::ScalarMul(-learning_rate, Affine1.delta_weight));
        Affine1.bias   = Matrix::Add(Affine1.bias  , Matrix::ScalarMul(-learning_rate, Affine1.delta_bias));
        Affine2.weight = Matrix::Add(Affine2.weight, Matrix::ScalarMul(-learning_rate, Affine2.delta_weight));
        Affine2.bias   = Matrix::Add(Affine2.bias  , Matrix::ScalarMul(-learning_rate, Affine2.delta_bias));
    }

    void forward_backward_test(const Matrix& x, const Matrix& t, float learning_rate)
    {
        //forward
        Matrix loss_value(1,1);
        loss_value = loss(x, t);
        assert(loss_value.elements[0] - 0.018638 < 1e-5);
        std::cout << "forward test passed\n";

        //backward
        Affine1.backward(RelU1.backward(Affine2.backward(SoftMaxWithLoss1.backward())));

        {
            Matrix test(Affine1.delta_weight.width, Affine1.delta_weight.height);
            Matrix test_ans(Affine1.delta_weight.width, Affine1.delta_weight.height);
            test = Affine1.delta_weight;
            load_data("data/grad_W1.tsv",test_ans);
            if_same_matrix(test, test_ans);
        }
        std::cout << "grad affin1_weight passed\n";

        {
            Matrix test(Affine1.delta_bias.width, Affine1.delta_bias.height);
            Matrix test_ans(Affine1.delta_bias.width, Affine1.delta_bias.height);
            test = Affine1.delta_bias;
            load_data("data/grad_b1.tsv",test_ans);
            if_same_matrix(test, test_ans);
        }
        std::cout << "grad affin1_bias passed\n";

        {
            Matrix test(Affine2.delta_weight.width, Affine2.delta_weight.height);
            Matrix test_ans(Affine2.delta_weight.width, Affine2.delta_weight.height);
            test = Affine2.delta_weight;
            load_data("data/grad_W2.tsv",test_ans);
            if_same_matrix(test, test_ans);
        }
        std::cout << "grad affin2_weight passed\n";

        {
            Matrix test(Affine2.delta_bias.width, Affine2.delta_bias.height);
            Matrix test_ans(Affine2.delta_bias.width, Affine2.delta_bias.height);
            test = Affine2.delta_bias;
            load_data("data/grad_b2.tsv",test_ans);
            if_same_matrix(test, test_ans);
        }
        std::cout << "grad affin2_bias passed\n";

        //update weight and bias
        Affine1.weight = Matrix::Add(Affine1.weight, Matrix::ScalarMul(-learning_rate, Affine1.delta_weight));
        Affine1.bias   = Matrix::Add(Affine1.bias  , Matrix::ScalarMul(-learning_rate, Affine1.delta_bias));
        Affine2.weight = Matrix::Add(Affine2.weight, Matrix::ScalarMul(-learning_rate, Affine2.delta_weight));
        Affine2.bias   = Matrix::Add(Affine2.bias  , Matrix::ScalarMul(-learning_rate, Affine2.delta_bias));


        {
            Matrix test(Affine1.delta_weight.width, Affine1.delta_weight.height);
            Matrix test_ans(Affine1.delta_weight.width, Affine1.delta_weight.height);
            test = Affine1.weight;
            load_data("data/updated_W1.tsv",test_ans);
            if_same_matrix(test, test_ans);
        }
        std::cout << "update affin1_weight passed\n";

        {
            Matrix test(Affine1.delta_bias.width, Affine1.delta_bias.height);
            Matrix test_ans(Affine1.delta_bias.width, Affine1.delta_bias.height);
            test = Affine1.bias;
            load_data("data/updated_b1.tsv",test_ans);
            if_same_matrix(test, test_ans);
        }
        std::cout << "update affin1_bias passed\n";

        {
            Matrix test(Affine2.delta_weight.width, Affine2.delta_weight.height);
            Matrix test_ans(Affine2.delta_weight.width, Affine2.delta_weight.height);
            test = Affine2.weight;
            load_data("data/updated_W2.tsv",test_ans);
            if_same_matrix(test, test_ans);
        }
        std::cout << "update affin2_weight passed\n";

        {
            Matrix test(Affine2.delta_bias.width, Affine2.delta_bias.height);
            Matrix test_ans(Affine2.delta_bias.width, Affine2.delta_bias.height);
            test = Affine2.bias;
            load_data("data/updated_b2.tsv",test_ans);
            if_same_matrix(test, test_ans);
        }
        std::cout << "update affin2_bias passed\n";
    }
};


int main()
{
    int   input_size = 784;
    int   hidden_size = 50;
    int   output_size = 10; //classification number
    int   batch_size  = 1;
    float learning_rate = 0.1;
    TwoLayerNetwork network(input_size, hidden_size, batch_size, output_size);

    Matrix h_weight1(input_size, hidden_size);
    Matrix h_bias1  (1, hidden_size);
    Matrix h_weight2(hidden_size, output_size);
    Matrix h_bias2  (1, output_size);
    Matrix h_input  (batch_size, input_size);
    Matrix h_teacher(batch_size, output_size);

    Matrix d_input  (batch_size, input_size, true);
    Matrix d_teacher(batch_size, output_size, true);

    load_data("data/W1.tsv", h_weight1);
    load_data("data/b1.tsv", h_bias1);
    load_data("data/W2.tsv", h_weight2);
    load_data("data/b2.tsv", h_bias2);
    load_data("data/input.tsv", h_input);
    load_data("data/teacher.tsv", h_teacher);

    d_input   = h_input;
    d_teacher = h_teacher;

    network.Affine1.weight = h_weight1;
    network.Affine1.bias   = h_bias1;
    network.Affine2.weight = h_weight2;
    network.Affine2.bias   = h_bias2;
    
    Matrix result(batch_size, output_size);
    result = network.predict(d_input);

    network.forward_backward_test(d_input, d_teacher, learning_rate);
}